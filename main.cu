#include "hip/hip_runtime.h"
/*
This project was created by Adrian Martin Diaz

It aims to create a simple converter to greyscale, taking an imput image 'sloth.png' (yes, the one of the labs) and
creating 'slothGreay.png' as an output, but this time using NPP instead of hardcoding directly into the GPU
*/ 

#include <hip/hip_runtime.h>
#include <npp.h>

#include <ImageIO.h>
#include <ImagesCPU.h>
#include <ImagesNPP.h>

#include <iostream>
#include <string>

//To keet things simple, everithing runs on the main function
int main()
{
    //We encapsulate everything in a try function to avoid errors (or at least to catch them)
    try
    {

        //We define the input and output name files
        std::string sFilename = "sloth.png";
        std::string sResultFilename = "slothGrey.png";

        //We load the input image knowing it is a RGB PNG
        npp::ImageCPU_8u_C4 oHostSrcColor;
        npp::loadImage(sFilename, oHostSrcColor); 
        npp::ImageNPP_8u_C4 oDeviceSrcColor(oHostSrcColor);

        //We declare the gray output
        npp::ImageNPP_8u_C1 oDeviceDstGray(oDeviceSrcColor.size());

        //This is the main body of the script, the function itself
        NPP_CHECK_NPP(nppiBGRAToGray_8u_C4C1R(
                            oDeviceSrcColor.data(), oDeviceSrcColor.pitch(),
                            oDeviceDstGray.data(), oDeviceDstGray.pitch(),
                            oDeviceDstGray.size()));


        //The host output file is set
        npp::ImageCPU_8u_C1 oHostFinal(oDeviceDstGray.size());
        oDeviceDstGray.copyTo(oHostFinal.data(), oHostFinal.pitch());

        //We save the image
        saveImage(sResultFilename, oHostFinal);
        std::cout << "Saved image: " << sResultFilename << std::endl;

        //We free memory
        nppiFree(oDeviceSrcColor.data());
        nppiFree(oDeviceDstGray.data());

        exit(EXIT_SUCCESS);
    }
    //Some catch errors, it is not compulsory but it is a good practice I saw in the labs
    catch (npp::Exception &rException)
    {
        std::cerr << "Program error! The following exception occurred: \n";
        std::cerr << rException << std::endl;
        std::cerr << "Aborting." << std::endl;

        exit(EXIT_FAILURE);
    }
    catch (...)
    {
        std::cerr << "Program error! An unknown type of exception occurred. \n";
        std::cerr << "Aborting." << std::endl;

        exit(EXIT_FAILURE);
        return -1;
    }

    return 0;
}